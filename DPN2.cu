
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <math.h>
#include <string.h>

#define BLOCK_WIDTH 128
#define MAX_GRID_WIDTH 65535

bool debug = false;

//print a list of numbers
void Printv(int n, double * l){
	if(n > 10) n = 10;
	for(int i = 0; i < n-1; ++i) printf("%.0f, ", l[i]);
	printf("%.0f\n", l[n-1]);
}
void Printv(int n, unsigned long long int * l){
	if(n > 10) n = 10;
	for(int i = 0; i < n-1; ++i) printf("%llu, ", l[i]);
	printf("%llu\n", l[n-1]);
}
__device__ void Printv_d(int n, double * l){
	if(n > 10) n = 10;
	for(int i = 0; i < n-1; ++i) printf("%.0f, ", l[i]);
	printf("%.0f\n", l[n-1]);
}
__device__ void Printv_d(int n, unsigned long long int * l){
	if(n > 10) n = 10;
	for(int i = 0; i < n-1; ++i) printf("%llu, ", l[i]);
	printf("%llu\n", l[n-1]);
}

bool cudaErrorCheck(){
	hipError_t hipError_t = hipGetLastError();
	if(hipError_t != hipSuccess){
		printf("KERNEL ERROR!\n");
		printf(hipGetErrorName(hipError_t));
		printf("\n");
		printf(hipGetErrorString(hipError_t));
		printf("\n\n");
		return true;
	}
	return false;
}

//obtain a global linearized index, unique to all threads
__device__ unsigned long long int GetGlobalIdx_d(){
	unsigned long long int bID;
	bID = 	gridDim.x * gridDim.y * blockIdx.z +
		gridDim.x * blockIdx.y +
		blockIdx.x;
			
	unsigned long long int gID;
	gID = bID * blockDim.x + threadIdx.x;
	return gID;
}

//obtain grid dimensions necessary to launch n threads
uint3 GetGridDimensions(unsigned long long int n){
	unsigned long long int totalBlocks = ceil((double)n / BLOCK_WIDTH); //total number of blocks necessary to complete the task
	
	if(totalBlocks < MAX_GRID_WIDTH)
		return make_uint3(totalBlocks, 1, 1);
	if(totalBlocks < (unsigned long long int) MAX_GRID_WIDTH * MAX_GRID_WIDTH)
		return make_uint3(MAX_GRID_WIDTH, ceil(totalBlocks / MAX_GRID_WIDTH), 1);
	unsigned long long int M3 = (unsigned long long int)MAX_GRID_WIDTH * MAX_GRID_WIDTH * MAX_GRID_WIDTH;
	if(totalBlocks < M3)
		return make_uint3(MAX_GRID_WIDTH, MAX_GRID_WIDTH, ceil(totalBlocks / ((unsigned long long int)MAX_GRID_WIDTH * MAX_GRID_WIDTH)));
	
	printf("ERROR - PROBLEM SIZE EXCEEDS MAXIMUM OPERATIONAL PARAMETERS. UNABLE TO DETERMINE GRID DIMENSIONS. (n = %llu)\n", n);
	return make_uint3(0, 0, 0);
}

//initialize some arrays to starting values
__global__ void Initialize_g(	unsigned long long int n,
				unsigned long long int s,
				double * l,
				double * filteredL,
				unsigned long long int * filteredNodes,
				bool debug
				){
	unsigned long long int gID = GetGlobalIdx_d();
	
	if(gID == 0) printf("Initializing...\n");
	
	if(gID < n){
		if(gID != s){
			l[gID] = INFINITY;
			filteredL[gID] = INFINITY;
		}
		else{
			l[gID] = 0;
			filteredL[gID] = 0;
		}
		filteredNodes[gID] = gID;
	}
	
	if(debug){
		__syncthreads();
		if(gID == 0) Printv_d(n, filteredNodes);
	}	
}
	
	
//partially reduce an array - each block determines the minimum of a subsection
//of the array, then writes that minimum into a result array
__global__ void PartialReduce_g(	unsigned long long int n,
					double * work,			//the array which will be reduced
					unsigned long long int * nodes,	//keeps track of which element of work 
									//belongs to which node
					bool debug
					){
	//block index
	unsigned long long int bID = 	gridDim.x * gridDim.y * blockIdx.z +
					gridDim.x * blockIdx.y +
					blockIdx.x;
					
	//global thread index
	unsigned long long int gID = GetGlobalIdx_d();
	
	//local thread index within its block
	int lID = threadIdx.x;
	
	if(debug){
		if(gID == 0){
			printf("Commencing partial reduction.\nSize: %llu\nInitial data (first 10 elements):\n  array: ", n);
			Printv_d(n, work);
			printf("  nodes: ");
			Printv_d(n, nodes);
		}
	}
	
	//assign shared memory
	extern __shared__ double shared[];
	double * sharedDist = shared;
	unsigned long long int * sharedNodes = (unsigned long long int *)&sharedDist[BLOCK_WIDTH];
	
	if(gID < n){
		sharedDist[lID] = work[gID];
		sharedNodes[lID] = nodes[gID];
	}
	__syncthreads();
	
	//reduce internally
	for(int stride = BLOCK_WIDTH/2; stride >= 1; stride /= 2){
		if(lID + stride < n){
			if(sharedDist[lID] > sharedDist[lID + stride]){
				sharedDist[lID] = sharedDist[lID + stride];
				sharedNodes[lID] = sharedNodes[lID + stride];
			}
		}
		__syncthreads();
	}
	
	//write results
	if(lID == 0){
		work[bID] = sharedDist[0];
		nodes[bID] = sharedNodes[0];
	}
	
	if(debug){
		__syncthreads();	
		if(gID == 0){
			printf("Partial reduction complete.\nResults:\n  array: ");
			Printv_d(5, work);
			printf("  nodes: ");
			Printv_d(5, nodes);
		}
	}
}

//expand a node - first mark it as expanded, then update the distance vector with possible new distances
__global__ void Update_g(	double * E,				//weighted adjacency matrix
				unsigned long long int n,		//total number of nodes
				unsigned long long int fN,		//total number of remaining nodes
				double * l,				//an array denoting the distance between
									//the starting node, and already expanded nodes
				unsigned long long int * path,		//path vector with information about which path
									//to take from s to a given node
				unsigned long long int u,		//node scheduled for expansion
				double * filteredL,			//l with expanded nodes filtered out
				unsigned long long int * filteredNodes,	//node indices with already expanded nodes
									//filtered out		
				bool debug
				){
	//global index
	unsigned long long int gID = GetGlobalIdx_d();	
	
	//filtered distance value
	double fL;
	
	//filtered node index
	unsigned long long int fNode;
	
	if(debug){
		if(gID == 0){
			printf("Commencing Update on node %llu.\nInput data:\n  l: ", u);
			Printv_d(5, l);
			printf("  filteredL: ");
			Printv_d(fN, filteredL);
			printf("  filteredNodes: ");
			Printv_d(fN, filteredNodes);
			printf("\n");
		}
	}
		
	if(gID < fN){
		//assign fNode and fL
		fL = filteredL[gID];
		fNode = filteredNodes[gID];
		//assign l[u] its final value
		if(fNode == u) l[u] = fL;
	}
		
	__syncthreads();

	//update the distance vector with possible new distances
	if(gID < fN){
		double dist = l[u] + E[u*n + fNode];
		if(dist < fL){
			filteredL[gID] = dist;
			path[fNode] = u;
			fL = dist;
		}
	}
	
	if(gID < fN){
		//eliminate u from the list of unexpanded nodes by filtering it out
		//move all further elements back one spot
		if(fNode > u){
			filteredL[gID-1] = fL;
			filteredNodes[gID-1] = fNode;
		}
	}
	
	if(debug){
		__syncthreads();
		if(gID == 0){
			printf("Update complete. Resulting data:\n  l:");
			Printv_d(n, l);
			printf("  filteredL: ");
			Printv_d(fN, filteredL);
			printf("  filteredNodes: ");
			Printv_d(fN, filteredNodes);
		}
	}
}

void Dijkstra(	double * E,			//weighted adjacency matrix
		unsigned long long int n,	//total number of nodes
		unsigned long long int s,	//starting node
		double * l,			//distance vector such that l[u] = d(s, u)
		unsigned long long int * path	//path vector such that the shortest path 
						//from s to u goes through p[u]
		){
	printf("Commencing Dijkstra...\n_______________________________\n\n\n\n\n\n\n\n\n\n\n\n\n\n\n\n\n\n\n\n");
	
	//define and allocate memory for various arrays
	double *E_d, *l_d, *workL_d, *filteredL_d;
	unsigned long long int *path_d, *workNodes_d, *filteredNodes_d;
	
	hipMalloc((void**) &E_d, n*n*sizeof(double));
	hipMalloc((void**) &l_d, n*sizeof(double));
	hipMalloc((void**) &workL_d, n*sizeof(double));
	hipMalloc((void**) &filteredL_d, n*sizeof(double));
	
	hipMalloc((void**) &path_d, n*sizeof(unsigned long long int));
	hipMalloc((void**) &workNodes_d, n*sizeof(unsigned long long int));
	hipMalloc((void**) &filteredNodes_d, n*sizeof(unsigned long long int));
	
	//transfer initial data to device memory
	hipMemcpy(E_d, E, n*n*sizeof(double), hipMemcpyHostToDevice);
	
	//initialize arrays
	printf("gridDim: (%u, %u, %u)\n", GetGridDimensions(n).x, GetGridDimensions(n).y, GetGridDimensions(n).z);
	Initialize_g<<<GetGridDimensions(n), BLOCK_WIDTH>>>(n, s, l_d, filteredL_d, filteredNodes_d, debug);
	hipDeviceSynchronize();
	
	unsigned long long int sharedMemSize;
	double * tempL = (double*)malloc(sizeof(double));
	unsigned long long int * current = (unsigned long long int*)malloc(sizeof(unsigned long long int));
	*current = s;
		
	//commence the algorithm
	//complete an initial update by expanding the starting node s
	printf("Starting initial update - expanding node %llu\n", s); 
	Update_g<<<GetGridDimensions(n), BLOCK_WIDTH>>>(	E_d,
								n,
								n,
								l_d,
								path_d,
								s,
								filteredL_d,
								filteredNodes_d,
								debug
								);
	hipDeviceSynchronize();
	if(cudaErrorCheck()) return;
	printf("Initial pdate completed successfully.\n");

	for(unsigned long long int i = 1; i < n; ++i){
		printf("Dijkstra iteration #%llu\n", i);
		
		//locate the next closest node via heavy reduction
		hipMemcpy(workL_d, filteredL_d, (n-i)*sizeof(double), hipMemcpyDeviceToDevice);
		hipMemcpy(workNodes_d, filteredNodes_d, (n-i)*sizeof(unsigned long long int), hipMemcpyDeviceToDevice);
		for(unsigned long long j = n-i; j>0; j /= BLOCK_WIDTH){
			sharedMemSize = BLOCK_WIDTH * (sizeof(double) + sizeof(unsigned long long int));
			PartialReduce_g<<<GetGridDimensions(j), BLOCK_WIDTH, sharedMemSize>>>(	j, 
												workL_d, 
												workNodes_d,
												debug
												);
			hipDeviceSynchronize();
			if(cudaErrorCheck()) return;
		}
		
		//reduction complete. closest node should now be workNodes_d[0]
		hipMemcpy(tempL, workL_d, sizeof(double), hipMemcpyDeviceToHost);
		hipMemcpy(current, workNodes_d, sizeof(unsigned long long int), hipMemcpyDeviceToHost);
		printf("Full reduction complete.\nClosest node: %llu; Distance: %.0f\n", *current, *tempL);
		printf("Expanding node %llu\n", *current);
		Update_g<<<GetGridDimensions(n-i), BLOCK_WIDTH>>>(	E_d,
									n,
									n-i,
									l_d,
									path_d,
									*current,
									filteredL_d,
									filteredNodes_d,
									debug
									);
		hipDeviceSynchronize();
		if(cudaErrorCheck()) return;
		printf("Update completed successfully.\n\n");
	}
	
	hipMemcpy(l, l_d, n*sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(path, path_d, n*sizeof(unsigned long long int), hipMemcpyDeviceToHost);
	
	printf("Algorithm complete. Results:\n  l: ");
	Printv(n, l);
	printf("  path: ");
	Printv(n, path);
	printf("\n");
}


int main(int argc, char *args[]){
	for(int i = 0; i < argc; ++i){
		if( strcmp(args[i], "-debug") == 0) debug = true;
	}

	unsigned long long int n = 5;
	
	double E[25] = {	0, 1, INFINITY, INFINITY, INFINITY,
				1, 0, 1, INFINITY, 10,
				INFINITY, 1, 0, INFINITY, 1,
				INFINITY, INFINITY, INFINITY, 0, 1,
				INFINITY, 10, 1, 1, 0
				};
	unsigned long long int s = 0;
	double * l = (double*)malloc(n*sizeof(double));
	unsigned long long int * path = (unsigned long long int*)malloc(n*sizeof(unsigned long long int));
	
	Dijkstra(E, n, s, l, path);
	
	return 0;
}
